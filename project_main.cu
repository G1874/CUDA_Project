#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <random>
#include <cmath>


typedef struct {
    float* weights;
    float* biases;
    float* inputs;
    float* outputs;
} layer_type;


float* load_data(const char* filename, int size)
{
    std::ifstream file(filename);
    if(!file.is_open())
    {
        std::cerr << "Error: file not found" << std::endl;
        return NULL;
    }

    float* data = new float[size];
    for(int i = 0; i < size; i++)
    {
        file >> data[i];
        if(file.peek() == ',')
            file.ignore();
    }
    file.close();

    return data;
}


void normalize_data(float* data, int size)
{
    for(int i = 0; i < size; i++)
    {
        data[i] = data[i] / 255.0;
    }
}


__global__ void relu(float* input, float* output)
{
    int batch_idx = blockIdx.x * blockDim.x;
    int idx = batch_idx * threadIdx.x;

    output[idx] = (input[idx] > 0.0f) ? input[idx] : 0.0f;
}


__global__ void reluDerivative(float* input, float* output)
{
    int batch_idx = blockIdx.x * blockDim.x;
    int idx = batch_idx * threadIdx.x;

    output[idx] = (input[idx] > 0.0f) ? 1.0f : 0.0f;
}


__global__ void softmax(float* input, float* output)
{
    int batch_idx = blockIdx.x * blockDim.x;
    int idx = batch_idx * threadIdx.x;

    extern __shared__ float sum[];

    atomicAdd(sum[batch_idx], expf(input[idx]));
    __syncthreads();

    output[idx] = expf(input[idx]) / sum[batch_idx];
}


__global__ void crossEntropyLoss(float* predictions, float* labels, float* loss)
{
    int batch_idx = blockIdx.x * blockDim.x;
    int idx = batch_idx * threadIdx.x;

    float l = -labels[idx] * logf(predictions[idx]);

    atomicAdd(loss[batch_idx], l);
}


__device__ void crossEntropyLossDerivative(float* predictions, float* labels, float* output)
{
    int batch_idx = blockIdx.x * blockDim.x;
    int idx = batch_idx * threadIdx.x;

    output[idx] = predictions[idx] - labels[idx];
}


__global__ void linearLayerForward(float* input, float* output, float* weights, float* biases,
                                   int inputSize, int outputSize, int batchSize)
{
    int batch_idx = blockIdx.x * blockDim.x;
    int neuron_idx = threadIdx.x;
    int output_idx = batch_idx + neuron_idx;
    int sample_idx = batch_idx * inputSize;

    float sum = 0.0f;
    for(int i=0; i<inputSize; i++)
    {
        sum += input[sample_idx + i] * weights[neuron_idx*inputSize + i];
    }
    
    output[output_idx] = sum + biases[neuron_idx];
}


__global__ void linearLayerBackward(float* error_grad, float* inputs, float* input_grad,
                                    float* weights_grad, float* biases_grad, int batchSize)
{
    int batch_idx = blockIdx.x * blockDim.x;
    int output_idx = batch_idx * threadIdx.x;
    int sample_idx = batch_idx * inputSize;

    for(int i=0; i<inputSize; i++)
    {
        weights_grad[output_idx*inputSize + i] = error_grad[output_idx] + inputs[sample_idx + i];
    } 
}


int main()
{
    const int input_size = 28 * 28;
    const int output_size = 10;
    const int hidden_size = 128;
    const int batch_size = 64;
    const int num_epochs = 10;
    const float learning_rate = 0.001f;

    float* x_train;
    float* y_train;
    float* x_test;
    float* y_test;

    x_train = load_data("dataset/x_train.txt", 60000 * input_size);
    y_train = load_data("dataset/y_train.txt", 60000);
    x_test = load_data("dataset/x_test.txt", 10000 * input_size);
    y_test = load_data("dataset/y_test.txt", 10000);

    normalize_data(x_train, 60000 * input_size);
    normalize_data(x_test, 10000 * input_size);

    //allocate host memory
    float* weights = new float[input_size * hidden_size + hidden_size * hidden_size + hidden_size * output_size];
    float* biases = new float[hidden_size + hidden_size + output_size];

    //initialize weights and biases
    std::mt19937 g(time(0));
    std::uniform_real_distribution<float> dist(0.0f, 1.0f);

    for(int i = 0; i < input_size * hidden_size + hidden_size * hidden_size + hidden_size * output_size; i++)
    {
        weights[i] = dist(g);
    }

    for(int i = 0; i < hidden_size + hidden_size + output_size; i++)
    {
        biases[i] = 0.0f;
    }

    //allocate device memory
    float *d_input, *d_hidden, *d_output, *d_weights, *d_biases;

    hipMalloc(&d_input, input_size * batch_size * sizeof(float));
    hipMalloc(&d_hidden, hidden_size * batch_size * sizeof(float));
    hipMalloc(&d_output, output_size * batch_size * sizeof(float));
    hipMalloc(&d_weights, input_size * hidden_size + hidden_size * hidden_size + hidden_size * output_size * sizeof(float));
    hipMalloc(&d_biases, hidden_size + hidden_size + output_size * sizeof(float));

    //copy data to device
    hipMemcpy(d_input, x_train, input_size * batch_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weights, weights, input_size * hidden_size + hidden_size * hidden_size + hidden_size * output_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_biases, biases, hidden_size + hidden_size + output_size * sizeof(float), hipMemcpyHostToDevice);

    //free host memory
    delete[] x_train;
    delete[] y_train;
    delete[] x_test;
    delete[] y_test;
    delete[] weights;
    delete[] biases;

    //free device memory
    hipFree(d_input);
    hipFree(d_hidden);
    hipFree(d_output);
    hipFree(d_weights);
    hipFree(d_biases);
}