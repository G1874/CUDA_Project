#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <random>
#include <cmath>


typedef struct {
    float* weights;
    float* biases;
    float* inputs;
    float* outputs;
    float* activation;
    // float* output_grad; ?
    // float* input_grad ?
    // float* weights_grad;
    // float* bias_grad;
} layer_type;


float* load_data(const char* filename, int size)
{
    std::ifstream file(filename);
    if(!file.is_open())
    {
        std::cerr << "Error: file not found" << std::endl;
        return NULL;
    }

    float* data = new float[size];
    for(int i = 0; i < size; i++)
    {
        file >> data[i];
        if(file.peek() == ',')
            file.ignore();
    }
    file.close();

    return data;
}


void normalize_data(float* data, int size)
{
    for(int i = 0; i < size; i++)
    {
        data[i] = data[i] / 255.0;
    }
}

void initialize_weights(float* weights, int size)
{
    std::mt19937 g(time(0));
    std::uniform_real_distribution<float> dist(0.0f, 1.0f);

    for(int i = 0; i < size; i++)
    {
        weights[i] = dist(g);
    }
}

void initialize_biases(float* biases, int size)
{
    for(int i = 0; i < size; i++)
    {
        biases[i] = 0.0f;
    }
}


__global__ void relu(float* input, float* output)
{
    int batch_idx = blockIdx.x * blockDim.x;
    int idx = batch_idx * threadIdx.x;

    output[idx] = (input[idx] > 0.0f) ? input[idx] : 0.0f;
}


__global__ void reluDerivative(float* input, float* output)
{
    int batch_idx = blockIdx.x * blockDim.x;
    int idx = batch_idx * threadIdx.x;

    output[idx] = (input[idx] > 0.0f) ? 1.0f : 0.0f;
}


__global__ void softmax(float* input, float* output)
{
    int batch_idx = blockIdx.x * blockDim.x;
    int idx = batch_idx * threadIdx.x;

    extern __shared__ float sum[];

    atomicAdd(sum[batch_idx], expf(input[idx]));
    __syncthreads();

    output[idx] = expf(input[idx]) / sum[batch_idx];
}


__global__ void crossEntropyLoss(float* predictions, float* labels, float* loss)
{
    int batch_idx = blockIdx.x * blockDim.x;
    int idx = batch_idx * threadIdx.x;

    float l = -labels[idx] * logf(predictions[idx]);

    atomicAdd(loss[batch_idx], l);
}


__global__ void crossEntropyLossDerivative(float* predictions, float* labels, float* error_grad)
{
    int batch_idx = blockIdx.x * blockDim.x;
    int idx = batch_idx * threadIdx.x;

    error_grad[idx] = predictions[idx] - labels[idx];
}


__global__ void linearLayerForward(layer_type layer, int inputSize, int outputSize, int batchSize)
{
    int batch_idx = blockIdx.x * blockDim.x;
    int neuron_idx = threadIdx.x;
    int output_idx = batch_idx + neuron_idx;
    int sample_idx = batch_idx * inputSize;

    float sum = 0.0f;
    for(int i=0; i<inputSize; i++)
    {
        sum += layer.inputs[sample_idx + i] * layer.weights[neuron_idx*inputSize + i];
    }
    
    layer.output[output_idx] = sum + layer.biases[neuron_idx];
}


__global__ void linearLayerBackward(layer_type layer, float* inputs, int inputSize, int outputSize, int batchSize)
{
    int batch_idx = blockIdx.x * blockDim.x;
    int output_idx = batch_idx * threadIdx.x;
    int sample_idx = batch_idx * inputSize;

    for(int i=0; i<inputSize; i++)
    {
        weights_grad[output_idx*inputSize + i] = layer.input_grad[output_idx] + inputs[sample_idx + i];
    } 
}


int main()
{
    const int input_size = 28 * 28;
    const int output_size = 10;
    const int hidden_size = 128;
    const int batch_size = 64;
    const int num_epochs = 10;
    const float learning_rate = 0.001f;

    float* x_train;
    float* y_train;
    float* x_test;
    float* y_test;

    // float* inputs = new float[input_size * batch_size];
    // float* predictions = new float[output_size * batch_size];

    x_train = load_data("dataset/x_train.txt", 60000 * input_size);
    y_train = load_data("dataset/y_train.txt", 60000);
    x_test = load_data("dataset/x_test.txt", 10000 * input_size);
    y_test = load_data("dataset/y_test.txt", 10000);

    normalize_data(x_train, 60000 * input_size);
    normalize_data(x_test, 10000 * input_size);

    //intialize host layers
    layer_type layer1, layer2, layer3;

    //layer 1
    layer1.weights = new float[input_size * hidden_size];
    layer1.biases = new float[hidden_size];

    //layer 2
    layer2.weights = new float[hidden_size * hidden_size];
    layer2.biases = new float[hidden_size];

    //layer 3
    layer3.weights = new float[hidden_size * output_size];
    layer3.biases = new float[output_size];


    //initialize weights and biases
    initialize_weights(layer1.weights, input_size * hidden_size);
    initialize_weights(layer2.weights, hidden_size * hidden_size);
    initialize_weights(layer3.weights, hidden_size * output_size);

    initialize_biases(layer1.biases, hidden_size);
    initialize_biases(layer2.biases, hidden_size);
    initialize_biases(layer3.biases, output_size);

    //allocate device memory
    // float* d_labels;

    layer_type d_layer1, d_layer2, d_layer3;

    hipMalloc((void**)&d_layer1.weights, input_size * hidden_size * sizeof(float));
    hipMalloc((void**)&d_layer1.biases, hidden_size * sizeof(float));
    hipMalloc((void**)&d_layer1.inputs, input_size * batch_size * sizeof(float));
    hipMalloc((void**)&d_layer1.outputs, hidden_size * batch_size * sizeof(float));
    hipMalloc((void**)&d_layer1.activation, hidden_size * batch_size * sizeof(float))

    hipMalloc((void**)&d_layer2.weights, hidden_size * hidden_size * sizeof(float));
    hipMalloc((void**)&d_layer2.biases, hidden_size * sizeof(float));
    hipMalloc((void**)&d_layer2.inputs, hidden_size * batch_size * sizeof(float));
    hipMalloc((void**)&d_layer2.outputs, hidden_size * batch_size * sizeof(float));
    hipMalloc((void**)&d_layer2.activation, hidden_size * batch_size * sizeof(float))

    hipMalloc((void**)&d_layer3.weights, hidden_size * output_size * sizeof(float));
    hipMalloc((void**)&d_layer3.biases, output_size * sizeof(float));
    hipMalloc((void**)&d_layer3.inputs, hidden_size * batch_size * sizeof(float));
    hipMalloc((void**)&d_layer3.outputs, output_size * batch_size * sizeof(float));
    hipMalloc((void**)&d_layer3.activation, output_size * batch_size * sizeof(float))


    //copy data to device
    hipMemcpy(d_layer1.inputs, x_train, input_size * batch_size * sizeof(float), hipMemcpyHostToDevice);
    // TODO:

    hipMemcpy(d_layer1.weights, layer1.weights, input_size * hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_layer1.biases, layer1.biases, hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_layer2.weights, layer2.weights, hidden_size * hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_layer2.biases, layer2.biases, hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_layer3.weights, layer3.weights, hidden_size * output_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_layer3.biases, layer3.biases, output_size * sizeof(float), hipMemcpyHostToDevice);



    
    for(int epoch = 0; epoch < 1; epoch++)
    {
        for(int i = 0; i < 60000 / batch_size; i++)
        {
            // Forward pass: input -> hidden
            linearLayerForward<<<batch_size, hidden_size>>>(d_layer1, input_size, hidden_size, batch_size);
            relu<<<batch_size, hidden_size>>>(d_layer1.outputs, d_layer1.activation);
            hipMemcpy(d_layer2.inputs, d_layer1.activation, hidden_size * batch_size * sizeof(float), hipMemcpyDeviceToDevice);

            // Forward pass: hidden -> hidden
            linearLayerForward<<<batch_size, hidden_size>>>(d_layer2, hidden_size, hidden_size, batch_size);
            relu<<<batch_size, hidden_size>>>(d_layer2.outputs, d_layer2.activation);
            hipMemcpy(d_layer3.inputs, d_layer2.activation, hidden_size * batch_size * sizeof(float), hipMemcpyDeviceToDevice);

            // Forward pass: hidden -> output
            linearLayerForward<<<batch_size, output_size>>>(d_layer3, hidden_size, output_size, batch_size);
            softmax<<<batch_size, output_size>>>(d_layer3.outputs, d_layer3.activation);

            // Loss TODO:
            // crossEntropyLoss<<<batch_size, output_size>>>(d_layer3.activation, d_labels, d_loss);
            // hipMemcpy(h_loss, d_loss, sizeof(float), hipMemcpyDeviceToHost);
            // printf("Batch: %d, Loss: %f", i, h_loss);

            // Backward pass: output -> hidden
            crossEntropyLossDerivative<<<batch_size, output_size>>>(d_layer3.activation, d_labels, d_layer3.input_grad); // Combined with softmax derivative
            linearLayerBackward<<<batch_size, output_size>>>(d_layer3, d_layer2.activation, hidden_size, output_size, batch_size);

            // Backward pass: hidden -> hidden
            reluDerivative<<<batch_size, hidden_size>>>(d_layer3.output_grad, d_layer2.input_grad);
            linearLayerBackward<<<batch_size, hidden_size>>>(d_layer2, d_layer1.activation, hidden_size, output_size, batch_size);

            // Backward pass: hidden -> input
            reluDerivative<<<batch_size, hidden_size>>>(d_layer2.output_grad, d_layer1.input_grad);
            linearLayerBackward<<<batch_size, hidden_size>>>(d_layer1, d_layer1.activation, hidden_size, output_size, batch_size);
        }
    }

    //free host memory
    delete[] x_train;
    delete[] y_train;
    delete[] x_test;
    delete[] y_test;

    //free device memory
    hipFree(d_layer1.weights);
    hipFree(d_layer1.biases);
    hipFree(d_layer1.inputs);
    hipFree(d_layer1.outputs);

    hipFree(d_layer2.weights);
    hipFree(d_layer2.biases);
    hipFree(d_layer2.inputs);
    hipFree(d_layer2.outputs);

    hipFree(d_layer3.weights);
    hipFree(d_layer3.biases);
    hipFree(d_layer3.inputs);
    hipFree(d_layer3.outputs);
}